#include "hip/hip_runtime.h"
// l1tex__average_t_sectors_per_request_pipe_lsu_mem_global_op_atom

#include "cuda_wrapper.h"
#include <iostream>
#include <vector>

__global__ void kernel_sum(hipTextureObject_t tex, const int height, const int width, float *res)
{
    const int x_index = blockIdx.x * blockDim.x + threadIdx.x;
    const int y_index = blockIdx.y * blockDim.y + threadIdx.y;

    if (x_index >= width || y_index >= height)
        return;

    atomicAdd(res, tex2D<float>(tex, x_index, y_index));
}

float cuMatSum(hipTextureObject_t tex, const int height, const int width, const dim3 grid, const dim3 block)
{
    float val = 0;
    float *res = NULL;
    CudaSafeCall(hipMalloc(&res, sizeof(float)));
    CudaSafeCall(hipMemcpy(res, &val, sizeof(float), hipMemcpyHostToDevice));

    kernel_sum<<<grid, block>>>(tex, height, width, res);
    CudaSafeCall(hipDeviceSynchronize());

    CudaSafeCall(hipMemcpy(&val, res, sizeof(float), hipMemcpyDeviceToHost));

    hipFree(res);
    return val;
}

void cuInitTexture(hipTextureObject_t &tex, const int width, const int height, hipArray *cuArray)
{
    std::vector<float> hData(1.f, width * height);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    CudaSafeCall(hipMallocArray(&cuArray, &channelDesc, width, height));
    CudaSafeCall(hipMemcpyToArray(cuArray, 0, 0, hData.data(), width * height * sizeof(float), hipMemcpyHostToDevice));


    hipResourceDesc texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));

    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = cuArray;

    hipTextureDesc texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));

    texDescr.normalizedCoords = true;
    texDescr.filterMode = hipFilterModeLinear;
    texDescr.addressMode[0] = hipAddressModeWrap;
    texDescr.addressMode[1] = hipAddressModeWrap;
    texDescr.readMode = hipReadModeElementType;

    CudaSafeCall(hipCreateTextureObject(&tex, &texRes, &texDescr, NULL));
}

void cuFreeTexture(hipTextureObject_t &tex, hipArray *cuArray)
{
    if (tex)
        CudaSafeCall(hipDestroyTextureObject(tex));
    if (cuArray)
        hipFreeArray(cuArray);
}

int main()
{
    const int width = 1024;
    const int height = 1024;

    hipTextureObject_t tex;
    hipArray *cuArray = NULL;

    cuInitTexture(tex, width, height, cuArray);

    dim3 grid(1, 1, 1);
    dim3 block(256, 1, 1);
    std::cout << cuMatSum(tex, 1, 256, grid, block) << std::endl;

    cuFreeTexture(tex, cuArray);

    return 0;
}
